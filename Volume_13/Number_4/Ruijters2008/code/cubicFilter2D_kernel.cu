#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2009, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.
\*--------------------------------------------------------------------------*/

#ifndef _CUBIC2D_KERNEL_H_
#define _CUBIC2D_KERNEL_H_

#include "bspline_kernel.cu"

//! Bilinearly interpolated texture lookup, using unnormalized coordinates.
//! This function merely serves as a reference for the bicubic versions.
//! @param tex  2D texture
//! @param x  unnormalized x texture coordinate
//! @param y  unnormalized y texture coordinate
template<class T, enum hipTextureReadMode mode>
__device__ float interpolate_bilinear(texture<T, 2, mode> tex, float x, float y)
{
	return tex2D(tex, x, y);
}

//! Bicubic interpolated texture lookup, using unnormalized coordinates.
//! Straight forward implementation, using 16 nearest neighbour lookups.
//! @param tex  2D texture
//! @param x  unnormalized x texture coordinate
//! @param y  unnormalized y texture coordinate
template<class T, enum hipTextureReadMode mode>
__device__ float interpolate_bicubic_simple(texture<T, 2, mode> tex, float x, float y)
{
	// transform the coordinate from [0,extent] to [-0.5, extent-0.5]
	const float2 coord_grid = make_float2(x - 0.5, y - 0.5);
	float2 index = floor(coord_grid);
	const float2 fraction = coord_grid - index;
	index.x += 0.5;  //move from [-0.5, extent-0.5] to [0, extent]
	index.y += 0.5;  //move from [-0.5, extent-0.5] to [0, extent]

	float result = 0.0;
	for (float y=-1; y < 2.5; y++)
	{
		float bsplineY = bspline(y-fraction.y);
		float v = index.y + y;
		for (float x=-1; x < 2.5; x++)
		{
			float bsplineXY = bspline(x-fraction.x) * bsplineY;
			float u = index.x + x;
			result += bsplineXY * tex2D(tex, u, v);
		}
	}
	return result;
}

//! Bicubic interpolated texture lookup, using unnormalized coordinates.
//! Fast implementation, using 4 trilinear lookups.
//! @param tex  2D texture
//! @param x  unnormalized x texture coordinate
//! @param y  unnormalized y texture coordinate
template<class T, enum hipTextureReadMode mode>
__device__ float interpolate_bicubic_fast(texture<T, 2, mode> tex, float x, float y)
{
	// transform the coordinate from [0,extent] to [-0.5, extent-0.5]
	const float2 coord_grid = make_float2(x - 0.5, y - 0.5);
	const float2 index = floor(coord_grid);
	const float2 fraction = coord_grid - index;
	float2 w0, w1, w2, w3;
	bspline_weights(fraction, w0, w1, w2, w3);

	const float2 g0 = w0 + w1;
	const float2 g1 = w2 + w3;
	const float2 h0 = (w1 / g0) - make_float2(0.5) + index;  //h0 = w1/g0 - 1, move from [-0.5, extent-0.5] to [0, extent]
	const float2 h1 = (w3 / g1) + make_float2(1.5) + index;  //h1 = w3/g1 + 1, move from [-0.5, extent-0.5] to [0, extent]

	// fetch the four linear interpolations
	float tex00 = tex2D(tex, h0.x, h0.y);
	float tex10 = tex2D(tex, h1.x, h0.y);
	float tex01 = tex2D(tex, h0.x, h1.y);
	float tex11 = tex2D(tex, h1.x, h1.y);

	// weigh along the y-direction
	tex00 = lerp(tex01, tex00, g0.y);
	tex10 = lerp(tex11, tex10, g0.y);

	// weigh along the x-direction
	return lerp(tex10, tex00, g0.x);
}


#endif // _CUBIC3D_KERNEL_H_
