#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2009, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.
\*--------------------------------------------------------------------------*/

#ifndef _CUDA_BSPLINE_H_
#define _CUDA_BSPLINE_H_

#include "cutil_math_bugfixes.h"
#include "math_func.cu"

// Cubic B-spline function
// The 3rd order Maximal Order and Minimum Support function, that it is maximally differentiable.
inline __device__ float bspline(float t)
{
	t = fabs(t);
	const float a = 2.0 - t;

	if (t < 1.0) return 2.0/3.0 - 0.5*t*t*a;
	else if (t < 2.0) return a*a*a / 6.0;
	else return 0.0;
}

// Inline calculation of the bspline weights, without conditional statements
template<class T> inline __device__ void bspline_weights(T fraction, T& w0, T& w1, T& w2, T& w3)
{
	const T one_frac = 1.0 - fraction;

	w0 = 1.0/6.0 * one_frac*one_frac*one_frac;
	w1 = 2.0/3.0 - 0.5 * fraction*fraction*(2.0-fraction);
	w2 = 2.0/3.0 - 0.5 * one_frac*one_frac*(2.0-one_frac);
	w3 = 1.0/6.0 * fraction*fraction*fraction;
}

#endif // _CUDA_BSPLINE_H_
